#include <gtest/gtest.h>
#include <gbs-cuda/basisfunctions.cuh>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
import math;
#include <gbs/basisfunctions.h>

namespace gbs
{
    struct f_gbs_cuda_basis_function
    {
        Id p_;
        device_vector<Real>::iterator knots_begin_;
        device_vector<Real>::iterator knots_end_;
        f_gbs_cuda_basis_function(
            Id p, 
            const device_vector<Real>::iterator &knots_begin,
            const device_vector<Real>::iterator &knots_end
        ) : p_{p}, knots_begin_{knots_begin}, knots_end_{knots_end} {}
        __device__
        auto operator()(Real u)
        {
            return basis_function(
                p_, 
                u, 
                knots_begin_, 
                knots_end_
            );
        }
    };
}


TEST(gbs_cuda, basis_function)
{
    using namespace gbs;

    const size_t p = 3;
    const size_t n = 100;
    std::vector<Real> knots_h
    {
        0., 0., 0.,
        0.3,
        0.7,
        1., 1., 1.
    };

    auto u_h = make_range<Real>(0., 1., n);
    std::vector<Real> eval_h(n);
    std::transform(
        u_h.begin(), u_h.end(),
        eval_h.begin(),
        [&](Real u){
            return basis_function(
                u, 
                std::begin(knots_h), 
                p, 
                std::end(knots_h)
            );
        }
    );


    device_vector<Real> u_d{ u_h };
    device_vector<Real> knots_d{knots_h};
    device_vector<Real> eval_d(n);

    thrust::transform(
        u_d.begin(), u_d.end(),
        eval_d.begin(),
        f_gbs_cuda_basis_function(p,knots_d.begin(), knots_d.end())
    );

    for(int i{}; i < n ; i++)
    {
        ASSERT_NEAR( eval_d[i] , eval_h[i], 1e-6 );
    }

}